#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUMS 4900

#define XMIN 0.
#define XMAX 3.
#define YMIN 0.
#define YMAX 3.

#define Z00	0.
#define Z10	1.
#define Z20	0.
#define Z30	0.

#define Z01	1.
#define Z11	6.
#define Z21	1.
#define Z31	0.

#define Z02	0.
#define Z12	1.
#define Z22	0.
#define Z32	4.

#define Z03	3.
#define Z13	2.
#define Z23	3.
#define Z33	3.

__device__ float Height(int iu, int iv);
__device__ float Height(int iu, int iv)// iu,iv = 0 .. NUMS-1
{
	float u = (float)iu / (float)(NUMS - 1);
	float v = (float)iv / (float)(NUMS - 1);

	// the basis functions:

	float bu0 = (1. - u) * (1. - u) * (1. - u);
	float bu1 = 3. * u * (1. - u) * (1. - u);
	float bu2 = 3. * u * u * (1. - u);
	float bu3 = u * u * u;

	float bv0 = (1. - v) * (1. - v) * (1. - v);
	float bv1 = 3. * v * (1. - v) * (1. - v);
	float bv2 = 3. * v * v * (1. - v);
	float bv3 = v * v * v;

	// finally, we get to compute something:

	float height = bu0 * (bv0*Z00 + bv1*Z01 + bv2*Z02 + bv3*Z03)
		+ bu1 * (bv0*Z10 + bv1*Z11 + bv2*Z12 + bv3*Z13)
		+ bu2 * (bv0*Z20 + bv1*Z21 + bv2*Z22 + bv3*Z23)
		+ bu3 * (bv0*Z30 + bv1*Z31 + bv2*Z32 + bv3*Z33);

	return height;
}


__global__ void bezierVolumeKernel(float *dev_volume) {
    extern __shared__ float partialSum[];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    float myPartialSum = 0.0f;
    float fullTileArea = (((XMAX - XMIN) / (float)(NUMS - 1))  *  ((YMAX - YMIN) / (float)(NUMS - 1))); // 假設全尺寸格點區域面積為1
    
    if (i < NUMS * NUMS) {
        int iu = i % NUMS;
        int iv = i / NUMS;
        
        // Corner...1/4 Area
        if (((iv == 0) && (iu == 0)) || ((iv == 0) && (iu == (NUMS - 1))) ||
            ((iv == (NUMS - 1)) && (iu == 0)) || ((iv == (NUMS - 1)) && (iu == (NUMS - 1)))) {
            myPartialSum = Height(iu, iv) * 0.25f * fullTileArea;
        }
        // Half size tile
        else if (((iv == 0) && ((iu != 0) || (iu != (NUMS - 1)))) ||
                 ((iv == (NUMS - 1)) && ((iu != 0) || (iu != (NUMS - 1)))) ||
                 ((iu == 0) && ((iv != 0) || (iv != (NUMS - 1)))) ||
                 ((iu == (NUMS - 1)) && ((iv != 0) || (iv != (NUMS - 1))))) {
            myPartialSum = Height(iu, iv) * 0.5f * fullTileArea;
        }
        // Full size tile
        else {
            myPartialSum = Height(iu, iv) * fullTileArea;
        }
        
        partialSum[tid] = myPartialSum;
        __syncthreads();
    }
    
    // 使用平行化減法樹來求和
    unsigned int stride = blockDim.x / 2;
    while (stride > 0) {
        if (tid < stride) {
            partialSum[tid] += partialSum[tid + stride];
        }
        __syncthreads();
        stride /= 2;
    }
    
    if (tid == 0) {
        atomicAdd(dev_volume, partialSum[0]);
    }
}

int main() {

    printf("computing with cuda\n");

    float *dev_volume;
    hipMalloc(&dev_volume, sizeof(float));
    hipMemset(dev_volume, 0, sizeof(float));
    
    dim3 blockSize(1024);
    dim3 gridSize((NUMS * NUMS + blockSize.x - 1) / blockSize.x);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    bezierVolumeKernel<<<gridSize, blockSize, blockSize.x * sizeof(float)>>>(dev_volume);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop); 
    // Compute execution time
    float gpu_time_use;
    hipEventElapsedTime(&gpu_time_use, start, stop);
    gpu_time_use = gpu_time_use / 1000.0;

    float volume = 0.0f;
    hipMemcpy(&volume, dev_volume, sizeof(float), hipMemcpyDeviceToHost);
    
    printf("Performance = %8.2lf MegaHeights/Sec\n", (NUMS * NUMS) / gpu_time_use / 1000000);
    printf("Total time :%8.4f seconds\n", gpu_time_use);
    printf("Total Volume of bezier surface:  %f\n", volume);
    
    hipFree(dev_volume);
    
    return 0;
}